#include "hip/hip_runtime.h"
#include "RecoLocalCalo/HGCalRecAlgos/interface/BinnerGPU.h"
#include "HeterogeneousCore/CUDAUtilities/interface/GPUVecArray.h"
#include <math.h>

#include "RecoLocalCalo/HGCalRecAlgos/interface/GPUHist2D.h"


namespace BinnerGPU {


  __global__ void kernel_compute_histogram(RecHitGPU*dInputData, Histo2D *dOutputData, const size_t numRechits) {

    size_t rechitLocation = blockIdx.x * blockDim.x + threadIdx.x;

    if(rechitLocation >= numRechits)
        return;

    float eta = dInputData[rechitLocation].eta;
    float phi = dInputData[rechitLocation].phi;
    unsigned int index = dInputData[rechitLocation].index;
   
    dOutputData->fillBinGPU(eta, phi, index);

  }


  float minEta = 1.6;
  float maxEta = 3.0;
  float minPhi = -M_PI;
  float maxPhi = M_PI;

//  std::shared_ptr<int> 
  Histo2D computeBins(std::vector<RecHitGPU> layerData) {
    Histo2D hOutputData(minEta, maxEta, minPhi, maxPhi);

    // Allocate memory and put data into device
    Histo2D *dOutputData;
    RecHitGPU* dInputData;
    hipMalloc(&dOutputData, sizeof(Histo2D));
    hipMalloc(&dInputData, sizeof(RecHitGPU)*layerData.size());
    hipMemcpy(dInputData, layerData.data(), sizeof(RecHitGPU)*layerData.size(), hipMemcpyHostToDevice);
    hipMemset(dOutputData, 0x00, sizeof(Histo2D));
    hipMemcpy(dOutputData, &hOutputData, sizeof(Histo2D), hipMemcpyHostToDevice);
  
    // Call the kernel
    const dim3 blockSize(1024,1,1);
    const dim3 gridSize(ceil(layerData.size()/1024.0),1,1);
    kernel_compute_histogram <<<gridSize,blockSize>>>(dInputData, dOutputData, layerData.size());

    // Copy result back!
    hipMemcpy(dOutputData, &hOutputData, sizeof(Histo2D), hipMemcpyDeviceToHost);

    // Free all the memory
    hipFree(dOutputData);
    hipFree(dInputData);

    
    return hOutputData;
  }

}
